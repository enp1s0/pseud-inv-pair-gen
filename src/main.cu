#include "hip/hip_runtime.h"
#include <iostream>
#include <cutf/memory.hpp>
#include <cutf/cublas.hpp>
#include <cutf/cusolver.hpp>
#include <cutf/hiprand.hpp>

#include <matfile/matfile.hpp>

template <class T>
std::string get_dtype_name_str();
template <>
std::string get_dtype_name_str<float >() {return "sp";}
template <>
std::string get_dtype_name_str<double>() {return "dp";}

template <class T>
__global__ void V_x_rS_x_Ut_kernel(
		const std::size_t m,
		const std::size_t n,
		const std::size_t r,
		T* const ra_ptr, const std::size_t lda,
		const T* const u_ptr, const std::size_t ldu,
		const T* const s_ptr,
		const T* const vt_ptr, const std::size_t ldvt
		) {
	const auto tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= m * n) {
		return;
	}

	const auto mi = tid % m;
	const auto ni = tid / m;

	T ra = 0;
	for (unsigned ri = 0; ri < r; ri++) {
		ra += vt_ptr[ldvt * mi + ri] * (1 / s_ptr[ri]) * u_ptr[ldu * ri + ni];
	}
	ra_ptr[mi + ni * lda] = ra;
}

template <class T>
void V_x_rS_x_Ut(
		const std::size_t m,
		const std::size_t n,
		const std::size_t r,
		T* const ra_ptr, const std::size_t lda,
		const T* const u_ptr, const std::size_t ldu,
		const T* const s_ptr,
		const T* const vt_ptr, const std::size_t ldvt
		) {
	const auto block_size = 256u;
	const auto grid_size= (m * n + block_size - 1) / block_size;

	V_x_rS_x_Ut_kernel<T>
		<<<grid_size, block_size>>>(
				m, n, r,
				ra_ptr, lda,
				u_ptr, ldu,
				s_ptr,
				vt_ptr, ldvt
				);
}

template <class T>
void generate_matrix_pair(
		const std::size_t m,
		const std::size_t n,
		const std::uint64_t seed,
		const bool check = false
		) {
	const auto matrix_id = m * 100000000lu + n * 1000lu + seed;
	std::stringstream ss;
	ss << std::hex << matrix_id;
	const std::string matrix_id_hex = ss.str();
	const std::string file_name_stem = matrix_id_hex + "-" + get_dtype_name_str<T>() + "-m" + std::to_string(m) + "-n" + std::to_string(n) + "-seed" + std::to_string(seed) + ".matrix";
	const std::string inv_file_name_stem = matrix_id_hex + "-" + get_dtype_name_str<T>() + "-inv-m" + std::to_string(n) + "-n" + std::to_string(m) + "-seed" + std::to_string(seed) + ".matrix";

	{
		auto d_mat = cutf::memory::get_device_unique_ptr<T>(m * n);
		auto h_mat = cutf::memory::get_host_unique_ptr  <T>(m * n);


		auto cugen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_MT19937);
		CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*cugen.get(), seed));

		CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), d_mat.get(), m * n));

		cutf::memory::copy(h_mat.get(), d_mat.get(), m * n);
		mtk::matfile::save_dense(
				m, n,
				h_mat.get(), m,
				file_name_stem
				);

		const std::size_t num_s = std::min(m, n);
		auto dS = cutf::memory::get_device_unique_ptr<T>(num_s);
		auto dU = cutf::memory::get_device_unique_ptr<T>(m * m);
		auto dVT = cutf::memory::get_device_unique_ptr<T>(n * n);
		auto hS = cutf::memory::get_host_unique_ptr<T>(num_s);
		auto hU = cutf::memory::get_host_unique_ptr<T>(num_s * m);
		auto hVT = cutf::memory::get_host_unique_ptr<T>(num_s * n);

		auto dInfo = cutf::memory::get_device_unique_ptr<int>(1);

		auto cusolver = cutf::cusolver::dn::get_handle_unique_ptr();

		int Lwork;
		CUTF_CHECK_ERROR(cutf::cusolver::dn::gesvd_buffer_size<T>(*cusolver.get(), m, n, &Lwork));

		auto dLwork_buffer = cutf::memory::get_device_unique_ptr<T>(Lwork);
		auto dRwork_buffer = cutf::memory::get_device_unique_ptr<T>(num_s - 1);

		CUTF_CHECK_ERROR(cutf::cusolver::dn::gesvd(
					*cusolver.get(),
					'S', 'S',
					m, n,
					d_mat.get(), m,
					dS.get(),
					dU.get(), m,
					dVT.get(), num_s,
					dLwork_buffer.get(),
					Lwork,
					dRwork_buffer.get(),
					dInfo.get()
					));

		V_x_rS_x_Ut(
				n, m, num_s,
				d_mat.get(), n,
				dU.get(), m,
				dS.get(),
				dVT.get(), num_s
				);

		cutf::memory::copy(h_mat.get(), d_mat.get(), m * n);
		mtk::matfile::save_dense(
				n, m,
				h_mat.get(), n,
				inv_file_name_stem
				);
	}
	std::printf("# Log\n");
	std::printf("A^t : %s\n", file_name_stem.c_str());
	std::printf("A   : %s\n", inv_file_name_stem.c_str());
	if (check) {
		auto mat_a_uptr = cutf::memory::get_host_unique_ptr<T>(m * n);
		auto mat_b_uptr = cutf::memory::get_host_unique_ptr<T>(m * n);
		mtk::matfile::load_dense(mat_b_uptr.get(), n, file_name_stem);
		mtk::matfile::load_dense(mat_a_uptr.get(), m, inv_file_name_stem);

		T o = 0;
#pragma omp marallel for collapse(2) reduction(+: o)
		for (unsigned i = 0; i < m; i++) {
			for (unsigned j = 0; j < m; j++) {
				T c = 0;
				for (unsigned k = 0; k < n; k++) {
					c += mat_a_uptr.get()[i + k * m] * mat_b_uptr.get()[k + j * n];
				}
				const auto diff = c - (i == j ? 1 : 0);
				o += diff * diff;
			}
		}
		o = std::sqrt(o / static_cast<T>(n));
		std::printf("orth: %e\n", o);
	}
}

int main(int argc, char** argv) {
	if (argc <= 4) {
		std::fprintf(stderr, "Usage: %s [N (N x N)] [dtype: fp32/fp64] [seed]\n", argv[0]);
		return 1;
	}

	const auto N = std::stoull(argv[1]);
	const auto dtype = std::string(argv[2]);
	const auto seed = std::stoull(argv[3]);

	if (dtype == "fp32") {
		generate_matrix_pair<float >(N, N, seed, true);
	} else if(dtype == "fp64") {
		generate_matrix_pair<double>(N, N, seed, true);
	} else {
		std::fprintf(stderr, "Error: Unknown dtype (%s)\n", dtype.c_str());
		return 1;
	}
}
